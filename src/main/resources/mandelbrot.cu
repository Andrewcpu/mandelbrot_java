#include "hip/hip_runtime.h"
#include <jni.h>
#include <hip/hip_runtime.h>
#include <math.h>
__global__ void mandelbrot(int *image, int width, int height, int max_iter,
                           double center_real, double center_imag, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    double aspect_ratio = (double)width / height;

    double real = ((x - width / 2.0) / (width * zoom)) * aspect_ratio + center_real;
    double imag = ((y - height / 2.0) / (height * zoom)) + center_imag;

    double zx = 0.0, zy = 0.0;
    int iteration = 0;

    while (zx * zx + zy * zy <= 4.0 && iteration < max_iter) {
        double temp = zx * zx - zy * zy + real;
        zy = 2.0 * zx * zy + imag;
        zx = temp;
        iteration++;
    }

    image[y * width + x] = iteration;
}

__global__ void mandelbrotWithColor(int *image, int width, int height, int max_iter,
                                    double center_real, double center_imag, double zoom) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    double aspect_ratio = (double)width / height;

    double real = ((x - width / 2.0) / (width * zoom)) * aspect_ratio + center_real;
    double imag = ((y - height / 2.0) / (height * zoom)) + center_imag;

    double zx = 0.0, zy = 0.0;
    int iteration = 0;
    while (zx * zx + zy * zy <= 4.0 && iteration < max_iter) {
        double temp = zx * zx - zy * zy + real;
        zy = 2.0 * zx * zy + imag;
        zx = temp;
        iteration++;
    }

    float hue = (float)(iteration % 256) / 255.0f;
    float saturation = 1.0f;
    float brightness = (iteration < max_iter) ? 1.0f : 0.0f;

    int i = (int)(hue * 6.0f);
    float f = (hue * 6.0f) - i;
    float p = brightness * (1.0f - saturation);
    float q = brightness * (1.0f - f * saturation);
    float t = brightness * (1.0f - (1.0f - f) * saturation);

    float r, g, b;
    switch (i % 6) {
        case 0: r = brightness; g = t; b = p; break;
        case 1: r = q; g = brightness; b = p; break;
        case 2: r = p; g = brightness; b = t; break;
        case 3: r = p; g = q; b = brightness; break;
        case 4: r = t; g = p; b = brightness; break;
        case 5: r = brightness; g = p; b = q; break;
    }

    int color = ((int)(r * 255) << 16) | ((int)(g * 255) << 8) | (int)(b * 255);

    image[y * width + x] = color;
}


extern "C" {
    JNIEXPORT void JNICALL Java_Mandelbrot_runMandelbrot(JNIEnv *env, jobject obj,
                                                     jintArray image, jint width,
                                                     jint height, jint maxIter,
                                                     jdouble centerReal, jdouble centerImag,
                                                     jdouble zoom) {
        jint *cImage = env->GetIntArrayElements(image, NULL);

        int *d_image;
        size_t size = width * height * sizeof(int);
        hipMalloc(&d_image, size);

        dim3 blockDim(32, 8); // Total threads per block = 256
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                     (height + blockDim.y - 1) / blockDim.y);


        mandelbrot<<<gridDim, blockDim>>>(d_image, width, height, maxIter, centerReal, centerImag, zoom);

        hipMemcpy(cImage, d_image, size, hipMemcpyDeviceToHost);

        hipFree(d_image);
        env->ReleaseIntArrayElements(image, cImage, 0);
    }
    JNIEXPORT void JNICALL Java_Mandelbrot_runMandelbrotWithColor(JNIEnv *env, jobject obj,
                                                                  jintArray image, jint width,
                                                                  jint height, jint maxIter,
                                                                  jdouble centerReal, jdouble centerImag,
                                                                  jdouble zoom) {
        jint *cImage = env->GetIntArrayElements(image, NULL);

        int *d_image;
        size_t size = width * height * sizeof(int);
        hipMalloc(&d_image, size);

        dim3 blockDim(32, 8); // Total threads per block = 256
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                     (height + blockDim.y - 1) / blockDim.y);

        mandelbrotWithColor<<<gridDim, blockDim>>>(d_image, width, height, maxIter,
                                                   centerReal, centerImag, zoom);

        hipMemcpy(cImage, d_image, size, hipMemcpyDeviceToHost);

        hipFree(d_image);
        env->ReleaseIntArrayElements(image, cImage, 0);
    }
}
